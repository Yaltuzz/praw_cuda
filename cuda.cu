
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <cstdlib>
#include <ctime>

#define min(a, b) (a < b ? a : b)

void mergesort(int*, int, dim3, dim3);
__global__ void gpu_mergesort(int*, int*, int, int, int, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(int*, int*, int, int, int);

__device__ void gpu_bottomUpMerge(int* source, int* dest, int start, int middle, int end) {
    int i = start;
    int j = middle;
    for (int k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

__global__ void gpu_mergesort(int* source, int* dest, int size, int width, int slices, dim3* threads, dim3* blocks) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int start = width*idx*slices, 
         middle, 
         end;

    for (int slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;
        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}


void mergesort(int* data, int size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    int* device_data;
    int* device_swap;
    dim3* device_threads;
    dim3* device_blocks;

    hipMalloc((void**) &device_data, size * sizeof(int));
    hipMalloc((void**) &device_swap, size * sizeof(int));

    // Copy from our input list into the first array
    hipMemcpy(device_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    //
    // Copy the thread / block info to the GPU as well
    //
    hipMalloc((void**) &device_threads, sizeof(dim3));
    hipMalloc((void**) &device_blocks, sizeof(dim3));

    hipMemcpy(device_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(device_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    int* A = device_data;
    int* B = device_swap;

    int nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        int slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, device_threads, device_blocks);
        hipDeviceSynchronize();
        // Switch the input / output arrays instead of copying them around
        A = A == device_data ? device_swap : device_data;
        B = B == device_data ? device_swap : device_data;
    }

    hipMemcpy(data, A, size * sizeof(int), hipMemcpyDeviceToHost);
}

int main(int argc, char** argv) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 128;
    blocksPerGrid.x = 64;

    int size = 1000;
    int* data = new int[size];
    std::srand(std::time(NULL)); 

    for (int i = 0; i < size; ++i) {
        data[i] = std::rand() % 150; 
    }

    mergesort(data, size, threadsPerBlock, blocksPerGrid);

    bool is_sorted = true;
    for (int i = 0; i < size - 1; i++) {
        if (data[i] > data[i + 1]) {
            is_sorted = false;
            break;
        }
    }
    std::cout << "Array is sorted: " << (is_sorted ? "true" : "false") << std::endl;
}
